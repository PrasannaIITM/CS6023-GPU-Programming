#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

#define MAX 1024

__global__ void dkernel(int m, int n, int *t, int *p, int * ans){
    __shared__ unsigned int firstFree, start;
    __shared__ int pRevMap[MAX];
    __shared__ int endTimes[MAX];

    int core = threadIdx.x;

    if(core == 0){

        firstFree = MAX;
        start = 0;

        for(int i = 0; i < MAX; i++){
            pRevMap[i] = -1;
            endTimes[i] = 0;
        }
    }

    __syncthreads();

    for(int ptr = 0;ptr < n;ptr++){
        firstFree = MAX;
        __syncthreads();

        if(endTimes[core] <= start)
            atomicMin(&firstFree, core);

        __syncthreads();


        if(pRevMap[p[ptr]] == -1){
            if(core == firstFree){
                pRevMap[p[ptr]] = firstFree;

                endTimes[firstFree] = start + t[ptr];

                ans[ptr] = start + t[ptr];
            }
        }
        else{
            if(core == pRevMap[p[ptr]]){
                if(start < endTimes[core]){
                    start = endTimes[core];
                }
                endTimes[core] = start + t[ptr];
                ans[ptr] = start + t[ptr];
            }

        }
        __syncthreads();

    }


}

//Complete the following function
void operations ( int m, int n, int *executionTime, int *priority, int *result )  {
    int *d_executionTime, *d_priority, *d_result;

    hipMalloc(&d_executionTime, n * sizeof(int));
    hipMalloc(&d_priority, n * sizeof(int));
    hipMalloc(&d_result, n * sizeof(int));

    hipMemcpy(d_executionTime, executionTime, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_priority, priority, n * sizeof(int), hipMemcpyHostToDevice);

    dkernel<<<1, m>>>(m, n, d_executionTime, d_priority, d_result);

    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);
}

int main(int argc,char **argv)
{
    int m,n;
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &m );      //scaning for number of cores
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of tasks
   
   //Taking execution time and priorities as input	
    int *executionTime = (int *) malloc ( n * sizeof (int) );
    int *priority = (int *) malloc ( n * sizeof (int) );
    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &executionTime[i] );
    }

    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &priority[i] );
    }

    //Allocate memory for final result output 
    int *result = (int *) malloc ( (n) * sizeof (int) );
    for ( int i=0; i<n; i++ )  {
        result[i] = 0;
    }
    
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    //==========================================================================================================
	

	operations ( m, n, executionTime, priority, result ); 
	
    //===========================================================================================================
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    //Total time of each task: Final Result
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", result[i]);
    }

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    
    free(executionTime);
    free(priority);
    free(result);
    
    
    
}